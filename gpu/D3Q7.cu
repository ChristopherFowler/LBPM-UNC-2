
#include <hip/hip_runtime.h>
/*
  Copyright 2013--2018 James E. McClure, Virginia Polytechnic & State University

  This file is part of the Open Porous Media project (OPM).
  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/
// GPU Functions for D3Q7 Lattice Boltzmann Methods

#define NBLOCKS 560
#define NTHREADS 128



__global__ void dvc_ScaLBL_Scalar_Pack_Many(int *list, int count, double *sendbuf, double *Data1, double *Data2, double *Data3, double *Data4, double *Data5, double *Data6, double *Data7, double *Data8, double *Data9, double *Data10, int N){
    //....................................................................................
    // Pack distribution q into the send buffer for the listed lattice sites
    // dist may be even or odd distributions stored by stream layout
    //....................................................................................
    int idx,n;
            idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx<count){
                n = list[idx];
                sendbuf[idx] = Data1[n];
                sendbuf[idx+count] = Data2[n];
                sendbuf[idx+2*count] = Data3[n];
                sendbuf[idx+3*count] = Data4[n];
                sendbuf[idx+4*count] = Data5[n];
                sendbuf[idx+5*count] = Data6[n];
                sendbuf[idx+6*count] = Data7[n];
                sendbuf[idx+7*count] = Data8[n];
                sendbuf[idx+8*count] = Data9[n];
                sendbuf[idx+9*count] = Data10[n];
        }
    }


__global__ void dvc_ScaLBL_Scalar_Unpack_Many(int *list, int count, double *recvbuf, double *Data1, double *Data2, double *Data3, double *Data4, double *Data5, double *Data6, double *Data7, double *Data8, double *Data9, double *Data10, int N){
    //....................................................................................
    // Pack distribution q into the send buffer for the listed lattice sites
    // dist may be even or odd distributions stored by stream layout
    //....................................................................................
    int idx,n;
       
           idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx<count){
                        n = list[idx];
                Data1[n] = recvbuf[idx];
                Data2[n] = recvbuf[idx+count];
                Data3[n] = recvbuf[idx+2*count];
                Data4[n] = recvbuf[idx+3*count];
                Data5[n] = recvbuf[idx+4*count];
                Data6[n] = recvbuf[idx+5*count];
                Data7[n] = recvbuf[idx+6*count];
                Data8[n] = recvbuf[idx+7*count];
                Data9[n] = recvbuf[idx+8*count];
                Data10[n] = recvbuf[idx+9*count];
        
    }
}




__global__  void dvc_ScaLBL_Scalar_Pack(int *list, int count, double *sendbuf, double *Data, int N){
    //....................................................................................
    // Pack distribution q into the send buffer for the listed lattice sites
    // dist may be even or odd distributions stored by stream layout
    //....................................................................................
    int idx,n;
    idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx<count){
        n = list[idx];
        sendbuf[idx] = Data[n];
    }
}
__global__  void dvc_ScaLBL_Scalar_Unpack(int *list, int count, double *recvbuf, double *Data, int N){
    //....................................................................................
    // Pack distribution q into the send buffer for the listed lattice sites
    // dist may be even or odd distributions stored by stream layout
    //....................................................................................
    int idx,n;
    idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx<count){
        n = list[idx];
        Data[n] = recvbuf[idx];
    }
}

__global__  void dvc_ScaLBL_PackDenD3Q7(int *list, int count, double *sendbuf, int number, double *Data, int N){
    //....................................................................................
    // Pack distribution into the send buffer for the listed lattice sites
    //....................................................................................
    int idx,n,component;
    idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx<count){
        for (component=0; component<number; component++){
            n = list[idx];
            sendbuf[idx*number+component] = Data[number*n+component];
            Data[number*n+component] = 0.0;    // Set the data value to zero once it's in the buffer!
        }
    }
}


__global__ void dvc_ScaLBL_UnpackDenD3Q7(int *list, int count, double *recvbuf, int number, double *Data, int N){
    //....................................................................................
    // Unack distribution from the recv buffer
    // Sum to the existing density value
    //....................................................................................
    int idx,n,component;
    idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx<count){
            for (component=0; component<number; component++){
            n = list[idx];
            Data[number*n+component] += recvbuf[idx*number+component];
        }
    }
}

__global__ void dvc_ScaLBL_D3Q7_Unpack(int q,  int *list,  int start, int count,
        double *recvbuf, double *dist, int N){
    //....................................................................................
    // Unpack distribution from the recv buffer
    // Distribution q matche Cqx, Cqy, Cqz
    // swap rule means that the distributions in recvbuf are OPPOSITE of q
    // dist may be even or odd distributions stored by stream layout
    //....................................................................................
    int n,idx;
    idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx<count){
        // Get the value from the list -- note that n is the index is from the send (non-local) process
        n = list[idx];
        // unpack the distribution to the proper location
        if (!(n<0)) { dist[q*N+n] = recvbuf[start+idx];
        //printf("%f \n",,dist[q*N+n]);
        }
    }
}

__global__ void dvc_ScaLBL_D3Q7_Init(char *ID, double *f_even, double *f_odd, double *Den, int Nx, int Ny, int Nz)
{
    int n,N;
    N = Nx*Ny*Nz;
    double value;
    char id;
    int S = N/NBLOCKS/NTHREADS + 1;
    for (int s=0; s<S; s++){
        //........Get 1-D index for this thread....................
        n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
        if (n<N){
           id = ID[n];
            if (id > 0){
                value = Den[n];
                f_even[n] = 0.3333333333333333*value;
                f_odd[n] = 0.1111111111111111*value;        //double(100*n)+1.f;
                f_even[N+n] = 0.1111111111111111*value;    //double(100*n)+2.f;
                f_odd[N+n] = 0.1111111111111111*value;    //double(100*n)+3.f;
                f_even[2*N+n] = 0.1111111111111111*value;    //double(100*n)+4.f;
                f_odd[2*N+n] = 0.1111111111111111*value;    //double(100*n)+5.f;
                f_even[3*N+n] = 0.1111111111111111*value;    //double(100*n)+6.f;
            }
            else{
                for(int q=0; q<3; q++){
                    f_even[q*N+n] = -1.0;
                    f_odd[q*N+n] = -1.0;
                }
                f_even[3*N+n] = -1.0;
            }
        }
    }
}

//*************************************************************************
__global__  void dvc_ScaLBL_D3Q7_Swap(char *ID, double *disteven, double *distodd, int Nx, int Ny, int Nz)
{
    int i,j,k,n,nn,N;
    // distributions
    double f1,f2,f3,f4,f5,f6;
    char id;
    N = Nx*Ny*Nz;

    int S = N/NBLOCKS/NTHREADS + 1;
    for (int s=0; s<S; s++){
        //........Get 1-D index for this thread....................
        n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;

        if (n<N ){
               id = ID[n];
            if (id > 0){
                //.......Back out the 3-D indices for node n..............
                k = n/(Nx*Ny);
                j = (n-Nx*Ny*k)/Nx;
                i = n-Nx*Ny*k-Nx*j;
                //........................................................................
                // Retrieve even distributions from the local node (swap convention)
                //        f0 = disteven[n];  // Does not particupate in streaming
                f1 = distodd[n];
                f3 = distodd[N+n];
                f5 = distodd[2*N+n];
                //........................................................................

                //........................................................................
                // Retrieve odd distributions from neighboring nodes (swap convention)
                //........................................................................
                nn = n+1;                            // neighbor index (pull convention)
                if (!(i+1<Nx))    nn -= Nx;            // periodic BC along the x-boundary
                //if (i+1<Nx){
                f2 = disteven[N+nn];                    // pull neighbor for distribution 2
                if (!(f2 < 0.0)){
                    distodd[n] = f2;
                    disteven[N+nn] = f1;
                }
                //}
                //........................................................................
                nn = n+Nx;                            // neighbor index (pull convention)
                if (!(j+1<Ny))    nn -= Nx*Ny;        // Perioidic BC along the y-boundary
                //if (j+1<Ny){
                f4 = disteven[2*N+nn];                // pull neighbor for distribution 4
                if (!(f4 < 0.0)){
                    distodd[N+n] = f4;
                    disteven[2*N+nn] = f3;
                }
                //........................................................................
                nn = n+Nx*Ny;                        // neighbor index (pull convention)
                if (!(k+1<Nz))    nn -= Nx*Ny*Nz;        // Perioidic BC along the z-boundary
                //if (k+1<Nz){
                f6 = disteven[3*N+nn];                // pull neighbor for distribution 6
                if (!(f6 < 0.0)){
                    distodd[2*N+n] = f6;
                    disteven[3*N+nn] = f5;
                }
            }
        }
    }
}

//*************************************************************************
__global__  void dvc_ScaLBL_D3Q7_Density(char *ID, double *disteven, double *distodd, double *Den,
        int Nx, int Ny, int Nz)
{
    char id;
    int n;
    double f0,f1,f2,f3,f4,f5,f6;
    int N = Nx*Ny*Nz;

    int S = N/NBLOCKS/NTHREADS + 1;
    for (int s=0; s<S; s++){
        //........Get 1-D index for this thread....................
        n = S*blockIdx.x*blockDim.x + s*blockDim.x + threadIdx.x;
        if (n<N){
            id = ID[n];
            if (id > 0 ){
                // Read the distributions
                f0 = disteven[n];
                f2 = disteven[N+n];
                f4 = disteven[2*N+n];
                f6 = disteven[3*N+n];
                f1 = distodd[n];
                f3 = distodd[N+n];
                f5 = distodd[2*N+n];
                // Compute the density
                Den[n] = f0+f1+f2+f3+f4+f5+f6;
            }
        }
    }
}

extern "C" void ScaLBL_D3Q7_Unpack(int q, int *list,  int start, int count, double *recvbuf, double *dist, int N){
    int GRID = count / 512 + 1;
    dvc_ScaLBL_D3Q7_Unpack <<<GRID,512 >>>(q, list, start, count, recvbuf, dist, N);
}

extern "C" void ScaLBL_Scalar_Pack(int *list, int count, double *sendbuf, double *Data, int N){
    int GRID = count / 512 + 1;
    dvc_ScaLBL_Scalar_Pack <<<GRID,512 >>>(list, count, sendbuf, Data, N);
}

extern "C" void ScaLBL_Scalar_Unpack(int *list, int count, double *recvbuf, double *Data, int N){
    int GRID = count / 512 + 1;
    dvc_ScaLBL_Scalar_Unpack <<<GRID,512 >>>(list, count, recvbuf, Data, N);
}
extern "C" void ScaLBL_PackDenD3Q7(int *list, int count, double *sendbuf, int number, double *Data, int N){
    int GRID = count / 512 + 1;
    dvc_ScaLBL_PackDenD3Q7 <<<GRID,512 >>>(list, count, sendbuf, number, Data, N);
}

extern "C" void ScaLBL_UnpackDenD3Q7(int *list, int count, double *recvbuf, int number, double *Data, int N){
    int GRID = count / 512 + 1;
    dvc_ScaLBL_UnpackDenD3Q7 <<<GRID,512 >>>(list, count, recvbuf, number, Data, N);
}

extern "C" void ScaLBL_D3Q7_Init(char *ID, double *f_even, double *f_odd, double *Den, int Nx, int Ny, int Nz){
    dvc_ScaLBL_D3Q7_Init <<<NBLOCKS,NTHREADS >>>(ID, f_even, f_odd, Den, Nx, Ny, Nz);
}

extern "C" void ScaLBL_D3Q7_Swap(char *ID, double *disteven, double *distodd, int Nx, int Ny, int Nz){
    dvc_ScaLBL_D3Q7_Swap <<<NBLOCKS,NTHREADS >>>(ID, disteven, distodd, Nx, Ny, Nz);
}

extern "C" void ScaLBL_D3Q7_Density(char *ID, double *disteven, double *distodd, double *Den,
                                        int Nx, int Ny, int Nz){
    dvc_ScaLBL_D3Q7_Density <<<NBLOCKS,NTHREADS >>>(ID, disteven, distodd, Den, Nx, Ny,  Nz);
}





extern "C" void ScaLBL_Scalar_Unpack_Many(int *list, int count, double *recvbuf, double *Data1, double *Data2, double *Data3, double *Data4, double *Data5, double *Data6, double *Data7, double *Data8, double *Data9, double *Data10, int N){
    int GRID = count / 512 + 1;
    dvc_ScaLBL_Scalar_Unpack_Many<<<GRID,512 >>>(list, count, recvbuf, Data1, Data2, Data3, Data4, Data5, Data6, Data7, Data8, Data9, Data10, N);

}

extern "C" void ScaLBL_Scalar_Pack_Many(int *list, int count, double *sendbuf, double *Data1, double *Data2, double *Data3, double *Data4, double *Data5, double *Data6, double *Data7, double *Data8, double *Data9, double *Data10, int N){
    int GRID = count / 512 + 1;
    dvc_ScaLBL_Scalar_Pack_Many<<<GRID,512 >>>(list, count, sendbuf,  Data1,  Data2,  Data3,  Data4,  Data5,  Data6, Data7, Data8,  Data9,  Data10, N);

}
